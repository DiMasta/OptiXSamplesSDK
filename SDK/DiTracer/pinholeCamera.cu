#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "helpers.h"

using namespace optix;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(rtObject, topObject, , );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

//**************************************************************************************************************************
//**************************************************************************************************************************

RT_PROGRAM void pinholeCamera() {
	float2 d = make_float2(launchIndex) / make_float2(launchDim) * 2.f - 1.f;
	float2 rayOrigin = eye;
	float2 rayDirection = normalize(d.x * U + d.y * V + W);

	optix::Ray ray = optix::make_Ray(rayOrigin, rayDirection, 0u, 1.e-4f, RT_DEFAULT_MAX);

	Payload payload;
	rtTrace(topObject, ray, payload);
}