#include "hip/hip_runtime.h"
#ifndef __PINHOLE_CAMERA_CU__
#define __PINHOLE_CAMERA_CU__

#include <optix_world.h>
#include "helpers.h"

using namespace optix;

struct RayState {
	float3 result;
};

//**************************************************************************************************************************
//**************************************************************************************************************************

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(rtObject, topObject, , );
rtBuffer<uchar4, 2> outputBuffer;

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

//**************************************************************************************************************************
//**************************************************************************************************************************

RT_PROGRAM void pinholeCamera() {
	float2 d = make_float2(launchIndex) / make_float2(launchDim) * 2.f - 1.f;
	float3 rayOrigin = eye;
	float3 rayDirection = normalize(d.x * U + d.y * V + W);

	optix::Ray ray = optix::make_Ray(rayOrigin, rayDirection, 0u, 1.e-4f, RT_DEFAULT_MAX);

	RayState rayState;
	rtTrace(topObject, ray, rayState);

	outputBuffer[launchIndex] = make_color(rayState.result);
}

#endif // __PINHOLE_CAMERA_CU__