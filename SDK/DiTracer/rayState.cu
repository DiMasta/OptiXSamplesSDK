#include "hip/hip_runtime.h"
#ifndef __RAYSTATE_CU__
#define __RAYSTATE_CU__

#include <optix_world.h>

struct RayState {
	float3 result;
};

//**************************************************************************************************************************
//**************************************************************************************************************************

rtDeclareVariable(RayState, rayState, rtPayload, );

#endif // __RAYSTATE_CU__