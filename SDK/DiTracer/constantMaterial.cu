#include "hip/hip_runtime.h"
#ifndef __CONSTANT_MATERIAL_CU__
#define __CONSTANT_MATERIAL_CU__

#include "raystate.cu"

rtDeclareVariable(float3, constantColor, , );

//**************************************************************************************************************************
//**************************************************************************************************************************

RT_PROGRAM void constantMaterialClosestHit() {
	rayState.result = constantColor;
}

#endif // __CONSTANT_MATERIAL_CU__