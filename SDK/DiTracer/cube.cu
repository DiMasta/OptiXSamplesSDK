#include "hip/hip_runtime.h"
#ifndef __CUBE_CU__
#define __CUBE_CU__

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtDeclareVariable(float3, cubeBottom, , );
rtDeclareVariable(float3, cubeTop, , );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

//**************************************************************************************************************************
//**************************************************************************************************************************

RT_PROGRAM void boundingBoxProgram(int primIdx, float result[6]) {
	Aabb* aabb = (Aabb*)result;

	aabb->set(cubeBottom, cubeTop);
}

//**************************************************************************************************************************
//**************************************************************************************************************************

RT_PROGRAM void intersectionProgram(int primIdx) {
	float3 origin = ray.origin;
	float3 direction = ray.direction;

	float3 t0 = (cubeBottom - origin) / direction;
	float3 t1 = (cubeTop - origin) / direction;

	float3 near = fminf(t0, t1);
	float3 far = fmaxf(t0, t1);

	float tmin = fmaxf(near);
	float tmax = fminf(far);

	if (tmin <= tmax) {
		bool checkSecond = true;

		if (rtPotentialIntersection(tmin)) {
			if (rtReportIntersection(0)) {
				checkSecond = false;
			}
		}

		if (checkSecond) {

			if (rtPotentialIntersection(tmax)) {
				rtReportIntersection(0);
			}
		}
	}
}

#endif // __CUBE_CU__