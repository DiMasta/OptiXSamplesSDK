#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3, cubeBottom, , );
rtDeclareVariable(float3, cubeTop, , );

//**************************************************************************************************************************
//**************************************************************************************************************************

RT_PROGRAM void boundingBoxProgram(int primIdx, float result[6]) {
	Aabb* aabb = (Aabb*)result;

	aabb->m_min = cubeBottom;
	aabb->m_max = cubeTop;
}

//**************************************************************************************************************************
//**************************************************************************************************************************

RT_PROGRAM void intersectionProgram(int primIdx) {

}